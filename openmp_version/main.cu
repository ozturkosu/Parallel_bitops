#include "hip/hip_runtime.h"
#include<iostream>
#include "mybitops.h"
#include <fstream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


//Project Fpr GPU classes

using namespace std;
#define DATA_GENERATION true

#define CHUNK_SIZE 3200


__device__ int word_type_device(size_t word1){

	if((word1 & 0x80000000) == 0)
		return 0; // literal word
	else{// fill word
		if((word1 & 0xc0000000) == 0xc0000000)
		return 2;//ones
		else
		return 1;//zeros

	}

}

__device__ int ismyfill_device(size_t word1){

	if((word1 & 0x80000000) == 0)
		return 0; // literal word
	else
		return 1; // fill word

}

__device__ size_t myBinarySearchGPU(size_t* arr, size_t l, size_t r, size_t x)
{
	if(x==0)
		return 0;
	if (r >= l)
	{
		size_t mid = l + (r - l)/2;
		// If the element is present at the middle
		// itself
		if (arr[mid] == x)
			return mid;
		// If element is smaller than mid, then
		// it can only be present in left subarray
		if (arr[mid] > x)
			return myBinarySearchGPU(arr, l, mid-1, x);
		// Else the element can only be present
		// in right subarray
		return myBinarySearchGPU(arr, mid+1, r, x);
	}

	// We reach here when element is not
	// present in array
	if(r<0)
		return 0;
	return r;

}

/*

__global__ void parallelAndDevice2(size_t * Vector1, int Vector1_size,  size_t * Vector2,
	int Vector2_size ,size_t * outVector, size_t vector1_bit_length, size_t vector2_bit_length)
{
		 int tid= blockIdx.x * blockDim.x + threadIdx.x ;

		 size_t min_bit_length = (vector1_bit_length<vector2_bit_length)?vector1_bit_length:vector2_bit_length;

		 size_t* word_lengths1 = new size_t[vec1_size]; //vector of length of the words
 		 size_t* prefix_sum1 = new size_t[vec1_size]; // prefix-sum generated form word_lengths1
 		 size_t pre_sum1 = 0;
 		 size_t vector1_bit_length = 0;

		 //Generate prefix sum and words size array for vector 1
		 for(int i = 0 ; i<vec1_size;i++)
		 {
			 size_t word = comp_bitset1[i];
			 int wt = Bitops.word_type(word);
			 size_t word_length = 31;
			 if(wt != 0)// if a fill word
				 word_length = word & 0x3fffffff;
			 word_lengths1[i] = word_length;
			 prefix_sum1[i] = pre_sum1+word_length;
			 pre_sum1 = prefix_sum1[i];
			 vector1_bit_length+=word_length;
		 }



		 size_t* word_lengths2 = new size_t[vec2_size]; //vector of length of the words
		 size_t* prefix_sum2 = new size_t[vec2_size];   // prefix-sum generated form word_lengths1
		 size_t pre_sum2 = 0;
		 size_t vector2_bit_length = 0;

		 for(int i = 0 ; i<vec2_size;i++)
		 {
			 size_t word = comp_bitset2[i];
			 int wt = Bitops.word_type(word);
			 size_t word_length = 31;
			 if(wt != 0)// if a fill word
				 word_length = word & 0x3fffffff;
			 word_lengths2[i] = word_length;
			 prefix_sum2[i] = pre_sum2+word_length;
			 // printf("%u\n",word_length);
			 pre_sum2 = prefix_sum2[i];
			 vector2_bit_length+=word_length;
		 }




}

*/

__global__ void parallelAndDevice(size_t * Vector1, int Vector1_size, size_t * prefixSum1, size_t * wordSize1  , size_t * Vector2,
	int Vector2_size, size_t * prefixSum2 , size_t * wordSize2 ,size_t * outVector, size_t vector1_bit_length, size_t vector2_bit_length)
{
	 //comp_bitset1_device[0]=1;
	 int tid= blockIdx.x * blockDim.x + threadIdx.x ;

	 size_t min_bit_length = (vector1_bit_length<vector2_bit_length)?vector1_bit_length:vector2_bit_length;

	 size_t *v1_ptr = &Vector1[0];
	 size_t *v2_ptr = &Vector2[0];


	 size_t b1 = tid*CHUNK_SIZE;
	 size_t b2 = (tid+1)*CHUNK_SIZE;

	 ////2.2 calc the word area
	 int w11 = myBinarySearchGPU(prefixSum1,0,Vector1_size-1,b1);
	 int w12 = myBinarySearchGPU(prefixSum1,0,Vector1_size-1,b2);

	 int w21 = myBinarySearchGPU(prefixSum2,0,Vector2_size-1,b1);
	 int w22 = myBinarySearchGPU(prefixSum2,0,Vector2_size-1,b2);

	 float res_size = ((w12-w11)<(w22-w21))?(w12-w11):(w22-w21);

	 size_t min_length = res_size * 1.01;//give 1% more capacity
	 //size_t *result_vector = new size_t[min_length] ;
	 size_t *result_vector = &outVector[0] ;
	 size_t *result_ptr = &result_vector[0] ;

	 size_t ones = 0, zeros = 0;
	 size_t it1 = w11;
	 size_t it2 = w21;
	 size_t word1 = Vector1[it1];
	 size_t word2 = Vector2[it2];
	 size_t count = 0;


	 while(it1 <= w12 && it2 <= w22) {
		 if(ismyfill_device(word1) && ismyfill_device(word2)) {  //both are fill words
			 size_t c1 = word1 & 0x3fffffff;         //get the len of fill word
			 size_t c2 = word2 & 0x3fffffff;         //get the len of fill word
			 size_t isone1 = word1 & 0x40000000;     //check if it is one
			 size_t isone2 = word2 & 0x40000000;     //check if it is one

			 if(c1 == c2) {
				 if(isone1 && isone2) {
					 //allZeros = false;
					 ones += c1;
					 if(zeros > 0) {
						 size_t n = 0x80000000 + zeros;
						 zeros = 0;
						 result_ptr[count] = n;
						 count++;
						 // result_vector.push_back(n);
					 }
				 } else {
					 zeros += c1;
					 if(ones > 0) {
						 size_t n = 0xc0000000 + ones;
						 ones = 0;
						 result_ptr[count] = n;
						 count++;
						 // result_vector.push_back(n);
					 }
				 }
				 word1 = Vector1[++it1];
				 word2 = Vector2[++it2];
			 } else if (c1 > c2) {
				 if(isone1 && isone2) {
					 //allZeros = false;
					 ones += c2;
					 if(zeros > 0) {
						 size_t n = 0x80000000 + zeros;
						 zeros = 0;
						 result_ptr[count] = n;
						 count++;
						 // result_vector.push_back(n);
					 }
				 } else {
					 zeros += c2;
					 if(ones > 0) {
						 size_t n = 0xc0000000 + ones;
						 ones = 0;
						 // result_vector.push_back(n);
					 }
				 }
				 if(isone1)
					 word1 = 0xc0000000 + (c1 - c2);
				 else
					 word1 = 0x80000000 + (c1 - c2);
				 word2 = Vector2[++it2];
			 } else { //c2 > c1
				 if(isone1 && isone2) {
					 //allZeros = false;
					 ones += c1;
					 if(zeros > 0) {
						 size_t n = 0x80000000 + zeros;
						 zeros = 0;
						 result_ptr[count] = n;
						 count++;
						 // result_vector.push_back(n);
					 }
				 } else {
					 zeros += c1;
					 if(ones > 0) {
						 size_t n = 0xc0000000 + ones;
						 ones = 0;
						 result_ptr[count] = n;
						 count++;
						 // result_vector.push_back(n);
					 }
				 }
				 if(isone2)
					 word2 = 0xc0000000 + (c2 - c1);
				 else
					 word2 = 0x80000000 + (c2 - c1);
				 word1 = Vector1[++it1];
			 }
		 } else if(ismyfill_device(word1) && !ismyfill_device(word2)) { //one fill, one literal
			 size_t c1 = word1 & 0x3fffffff; //lenth
			 size_t isone1 = word1 & 0x40000000;  //is one?

			 if(isone1) { //if one, depend on literal
				 if(ones > 0) {
					 size_t n = 0xc0000000 + ones;
					 //allZeros = false;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 if(zeros > 0) {
					 size_t n = 0x80000000 + zeros;
					 zeros = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 result_ptr[count] = word2;
				 count++;
				 // result_vector.push_back(word2);
			 } else { //id zero, generate a zero word
				 if(ones > 0) {
					 size_t n = 0xc0000000 + ones;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 zeros += 31;
			 }
			 if(c1 == 31){
				 word1 = Vector1[++it1];
			 }
			 else {
				 if(isone1)
					 word1 = 0xc0000000 + (c1 - (size_t)31);
				 else
					 word1 = 0x80000000 + (c1 - (size_t)31);
			 }
			 word2 = Vector2[++it2];
		 } else if(!ismyfill_device(word1) && ismyfill_device(word2)) { //one literal, one fill
			 size_t c2 = word2 & 0x3fffffff; //lenth
			 size_t isone2 = word2 & 0x40000000;  //is one?

			 if(isone2) { //if one, depend on literal
				 if(ones > 0) {
					 size_t n = 0xc0000000 + ones;
					 //allZeros = false;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 if(zeros > 0) {
					 size_t n = 0x80000000 + zeros;
					 zeros = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 //if (word1 != 0) //allZeros = false;
				 result_ptr[count] = word1;
				 count++;
				 // result_vector.push_back(word1);
			 } else { //id zero, generate a zero word
				 if(ones > 0) {
					 size_t n = 0xc0000000 + ones;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 zeros += 31;
			 }
			 if(c2 == 31)
				 word2 = Vector2[++it2];
			 else {
				 if(isone2)
					 word2 = 0xc0000000 + (c2 - (size_t)31);
				 else
					 word2 = 0x80000000 + (c2 - (size_t)31);
			 }
			 word1 = Vector1[++it1];
		 } else { //both are literal words
			 size_t num = word1 & word2;
			 if(num == 0) {
				 zeros += 31;
				 if(ones > 0) {
			 //allZeros = false;
					 size_t n = 0xc0000000 + ones;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
			 } else if (num == 0x7fffffff) {
				 ones += 31;
				 //allZeros = false;
				 if(zeros > 0) {
					 size_t n = 0x80000000 + zeros;
					 zeros = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
			 } else {
				 if(ones > 0) {
					 //allZeros = false;
					 size_t n = 0xc0000000 + ones;
					 ones = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n);
				 }
				 if(zeros > 0) {
					 size_t n = 0x80000000 + zeros;
					 zeros = 0;
					 result_ptr[count] = n;
					 count++;
					 // result_vector.push_back(n); //// freq access
				 }
				 //if (num != 0) //allZeros = false;
				 result_ptr[count] = num;
				 count++;
				 // result_vector.push_back(num);//// freq access
			 }
			 word1 = Vector1[++it1];
			 word2 = Vector2[++it2];
		 }
	 }
	 if(ones > 0) {
	 //allZeros = false;
		 size_t n = 0xc0000000 + ones;
		 ones = 0;
		 result_ptr[count] = n;
		 count++;
		 // result_vector.push_back(n);
	 }
	 if(zeros > 0) {
		 size_t n = 0x80000000 + zeros;
		 zeros = 0;
		 result_ptr[count] = n;
		 count++;
		 // result_vector.push_back(n);
	 }



}



int main(int argc, char** argv)
{
    cout<<"Usage: ./proj num_threads\n";
    cout<<"***************************************************************\n";
    int NUM_THREADS_OPENMP =atoi(argv[1]);


    vector<size_t> comp_bitset1;
    vector<size_t> comp_bitset2;

    mybitops Bitops;
    if(DATA_GENERATION)
    {
				cout<<"data generation*\n";
        size_t items = 20000000;
        float fill_percent = .001;
        boost::dynamic_bitset<> tempvector(items);
        cout<<"Data Generation"<<endl;
        for(size_t i=0; i<items*fill_percent; i++) {
            size_t rnd = rand()%items;
            while(tempvector[rnd]==1)
                rnd = rand()%items;
            tempvector[rnd] = 1;
        }

        cout<<"Data Compression"<<endl;
        clock_t t0 = clock();
        comp_bitset1 = Bitops.compressBitset(tempvector);

        clock_t t1 = clock();
        cout<<"compression time:"<< t1-t0<<endl;
        cout<<"vector size:"<< comp_bitset1.size()<<endl;
        comp_bitset2 = comp_bitset1;
        Bitops.save_vector(comp_bitset1,"./vec4", 1000);
    }
    else
    {
        comp_bitset1 = Bitops.load_vector("../data/lvec1");
        comp_bitset2 = Bitops.load_vector("../data/lvec1");
    }

    // clock_t t2 = clock();
    //Bitops.parallel_and(comp_bitset1,comp_bitset2,NUM_THREADS);
    // clock_t t3 = clock();
    // cout<<"parallel_and time:"<< t3-t2<<endl;

    clock_t t4 = clock();
    Bitops.logic_and_ref(&comp_bitset1,&comp_bitset2,0,comp_bitset1.size()-1,0,comp_bitset2.size()-1);
    clock_t t5 = clock();
    cout<<"seq_and time:"<< t5-t4<<endl;
    //pthread_exit(NULL);


		//Lets do it in Cuda version

		//thrust::device_vector

		thrust::device_vector<size_t>  comp_bitset1_device;
		thrust::device_vector<size_t>  comp_bitset2_device;

		comp_bitset1_device = comp_bitset1 ;
		comp_bitset2_device = comp_bitset2 ;


		//size_t* comp_dev_vect1 = thrust::raw_pointer_cast(&comp_bitset1_device) ;
		//size_t* comp_dev_vect2 = thrust::raw_pointer_cast(&comp_bitset2_device) ;


		//lets calculate prefix sum and word array for cuda calculation

		size_t vec1_size = comp_bitset1.size();
		size_t vec2_size = comp_bitset2.size();


		//Create word and prefix sum for vector 1

		size_t* word_lengths1 = new size_t[vec1_size]; //vector of length of the words
		size_t* prefix_sum1 = new size_t[vec1_size]; // prefix-sum generated form word_lengths1
		size_t pre_sum1 = 0;
		size_t vector1_bit_length = 0;

		hipEvent_t startEventPrefixSum, stopEventPrefixSum;




		//Generate prefix sum and words size array for vector 1
		for(int i = 0 ; i<vec1_size;i++)
		{
			size_t word = comp_bitset1[i];
			int wt = Bitops.word_type(word);
			size_t word_length = 31;
			if(wt != 0)// if a fill word
				word_length = word & 0x3fffffff;
			word_lengths1[i] = word_length;
			prefix_sum1[i] = pre_sum1+word_length;
			pre_sum1 = prefix_sum1[i];
			vector1_bit_length+=word_length;
		}


		//Create word and prefix sum for vector 2

		size_t* word_lengths2 = new size_t[vec2_size]; //vector of length of the words
		size_t* prefix_sum2 = new size_t[vec2_size];   // prefix-sum generated form word_lengths1
		size_t pre_sum2 = 0;
		size_t vector2_bit_length = 0;

		for(int i = 0 ; i<vec2_size;i++)
	  {
	    size_t word = comp_bitset2[i];
	    int wt = Bitops.word_type(word);
	    size_t word_length = 31;
	    if(wt != 0)// if a fill word
	      word_length = word & 0x3fffffff;
	    word_lengths2[i] = word_length;
	    prefix_sum2[i] = pre_sum2+word_length;
	    // printf("%u\n",word_length);
	    pre_sum2 = prefix_sum2[i];
	    vector2_bit_length+=word_length;
	  }


		hipEvent_t startEvent_kernel, stopEvent_kernel;

		size_t * Vector1 = (size_t*)malloc( vec1_size * sizeof(size_t)) ;
		size_t * Vector2 = (size_t*)malloc( vec2_size * sizeof(size_t)) ;

		size_t outVectorSize = (vec1_size < vec2_size)?vec1_size:vec2_size ;

		size_t * outVector = (size_t*)malloc(outVectorSize * sizeof(size_t) ) ;


		size_t * Vector1_device;
		size_t * Vector2_device;
		size_t * outVector_device;
		size_t * presum2_device ;
		size_t * presum1_device ;
		size_t * word_length_device ;
		size_t * word_length_device2 ;


		std::copy(comp_bitset1.begin(), comp_bitset1.end() , Vector1);
		std::copy(comp_bitset2.begin(), comp_bitset2.end() , Vector2);


		hipMalloc((void**) &Vector1_device , vec1_size * sizeof(size_t) );
		hipMalloc((void**) &Vector2_device , vec2_size * sizeof(size_t) );
		hipMalloc((void**) &outVector_device , outVectorSize * sizeof(size_t)) ;
		hipMalloc((void**) &presum1_device , vec1_size * sizeof(size_t)) ;
		hipMalloc((void**) &presum2_device , vec2_size * sizeof(size_t)) ;
		hipMalloc((void**) &word_length_device , vec1_size * sizeof(size_t)) ;
		hipMalloc((void**) &word_length_device2 , vec2_size * sizeof(size_t)) ;


		hipMemcpy(Vector1_device, Vector1 ,vec1_size * sizeof(size_t) , hipMemcpyHostToDevice ) ;
		hipMemcpy(Vector2_device, Vector2 ,vec2_size * sizeof(size_t) , hipMemcpyHostToDevice ) ;
		hipMemcpy(presum1_device, prefix_sum1 , vec1_size * sizeof(size_t) , hipMemcpyHostToDevice ) ;
		hipMemcpy(presum2_device, prefix_sum2 , vec2_size * sizeof(size_t) , hipMemcpyHostToDevice ) ;
		hipMemcpy(word_length_device , word_lengths1 , vec1_size * sizeof(size_t) ,  hipMemcpyHostToDevice  ) ;
		hipMemcpy(word_length_device2 , word_lengths2 , vec2_size * sizeof(size_t) ,  hipMemcpyHostToDevice  ) ;



		dim3 dimGrid(10000,1,1);
		dim3 dimBlock(128,1,1);


		hipEventCreate(&startEvent_kernel);
		hipEventCreate(&stopEvent_kernel) ;

		hipEventRecord(startEvent_kernel, 0) ;

		clock_t t_KERNEL1 = clock();

		parallelAndDevice<<<dimGrid, dimBlock>>>(Vector1_device , vec1_size, presum1_device, word_length_device,
		                Vector2_device , vec2_size ,  presum2_device , word_length_device2 , outVector_device, vector1_bit_length , vector2_bit_length) ;

		clock_t t_KERNEL2 = clock();

		cout<<"seq_and time:"<< t5-t4<<endl;

		hipEventRecord(stopEvent_kernel, 0) ;

		float timeForKernel;
		hipEventElapsedTime(&timeForKernel, startEvent_kernel, stopEvent_kernel) ;

		printf("  Time for  Kernel : %f\n",  timeForKernel);

		hipMemcpy(outVector , outVector_device , outVectorSize * sizeof(size_t) , hipMemcpyDeviceToHost) ;


		hipFree(Vector1_device);
		hipFree(Vector2_device);
		hipFree(outVector_device);
		hipFree(presum1_device);
		hipFree(presum2_device);
		hipFree(word_length_device);
		hipFree(word_length_device2) ;




    return 0;
}
